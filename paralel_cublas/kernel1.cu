#include <stdio.h>
#include <hip/hip_runtime.h>
#include "hipblas.h"

#include "common.h"

const int num_submatrix = 8;



void msplitm(char transa, char transb, unsigned long long m, unsigned long long n, unsigned long long k, float alpha, const float *A, int lda, const float *B, int ldb, float beta, float *C, int ldc)
{
    printf("entering msplitm \n");
    float* A_d;
    float* B_d;
    float* C_d;
    unsigned long long A_sz = m * k;
    unsigned long long B_sz = n * k;
    unsigned long long C_sz = m * n;
    unsigned long long MAX =  (unsigned long long )m* (unsigned long long) n / num_submatrix;

	
	MAX -= MAX % k;
	printf("MAX: %d\n", MAX);
	printf("B_sz: %d\n",B_sz);
	unsigned long long numSubMatrixB = B_sz / MAX;
	printf("SubmatriciesB: %d\n", numSubMatrixB);
	unsigned long long SMB_sz = B_sz / numSubMatrixB;
	printf("SMB_sz: %d\n", SMB_sz);
	unsigned long long subCols = B_sz / (numSubMatrixB * k);
	printf("subCols: %d\n", subCols);
	unsigned long long numSubMatrixA = A_sz / MAX;
	unsigned long long SMA_sz = A_sz / numSubMatrixA;
	unsigned long long subRows = A_sz / (numSubMatrixA * k);
	printf("subrows: %d\n", subRows);
	printf("SMA_sz: %d\n", SMA_sz);
	printf("submatriciesA: %d\n", numSubMatrixA);
	unsigned long long overflowA = m % subRows;
	unsigned long long overflowB = n % subCols;
	printf("overflowB: %d\n", overflowB);
	printf("overflowA: %d\n", overflowA);
	for(unsigned long long i = 0; i < numSubMatrixB + 1; ++i){
		if(overflowB == 0 && i == numSubMatrixB){
			break;
		}
		float* b = 0;
		float* temp3 = (float*) malloc( sizeof(float)*subCols * k );
		for(int j = 0; j < k; ++j){
			for(int x = 0; x < subCols; ++x){
				if(i * subCols + x < n){
					temp3[j * subCols + x] = B[j * n + (i*subCols + x)];
				}else{
					temp3[j *subCols + x] = 0;
				}
			}
		}
		hipMalloc((void**) &b, sizeof(float) * subCols * k);
		hipMemcpy(b, temp3, sizeof(float)*subCols*k, hipMemcpyHostToDevice);
		free(temp3);
		for(unsigned long long y = 0; y < numSubMatrixA + 1; ++y){
			if(overflowA == 0 && y == numSubMatrixA){
				break;
			}
			float * temp = (float*) malloc( sizeof(float)*subRows * k );
			for(int j = 0; j < subRows; ++j){
				for(int x = 0; x < k; ++x){
					if(y * subRows + j < m){
						temp[j * k + x] = A[y*subRows*k + j*k + x];
					}else{
						temp[j * k + x] = 0;
					}
				}			
			}
			float* a = 0;
			float* c = 0;
			hipMalloc((void**) &a, sizeof(float) * subRows * k);
			hipMalloc((void**) &c, sizeof(float) * subCols * subRows);
			hipMemcpy(a, temp, sizeof(float)*subRows*k, hipMemcpyHostToDevice);
			doMultiply2Matrices(subRows, k, a, k, subCols, b, c, alpha); 			
			hipMemcpy(temp, c, sizeof(float)*subRows*subCols, hipMemcpyDeviceToHost);
			if(i == numSubMatrixB && y == numSubMatrixA){
				copyElements(C, temp, subRows, subCols, m, n, y, i, overflowA, overflowB, beta);
			}else if(i == numSubMatrixB){
				copyElements(C, temp, subRows, subCols, m, n, y, i, 0, overflowB, beta);
			}else if(y == numSubMatrixA){
				copyElements(C, temp, subRows, subCols, m, n, y, i, overflowA, 0, beta);
			}else{
				copyElements(C, temp, subRows, subCols, m, n, y, i, 0, 0, beta);
			}
			free(temp);
			hipFree(a);
			hipFree(c);
		
		}
		
		hipFree(b);
	}
}






#include "hip/hip_runtime.h"
/**
*	Author : Rushikesh Gaidhani
*	Topic  : Matrix Multiplication on GPGPU using CUDA
*/

#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<hip/hip_runtime.h> 

hipEvent_t start, stop;  // using cuda events to measure time
float elapsed_time_ms;    // which is applicable for asynchronous code also

//Matrix multiplication kernel - thread specification
__global__ void MatrixMulKernel(float *Md, float *Nd, float *Pd, int Width)
{
    //2D Thread ID
    int tx = threadIdx.x + blockIdx.x*blockDim.x;
    int ty = threadIdx.y + blockIdx.y*blockDim.y;
    
    //Pvalue stores the Pd element that is computed by the thread
    float Pvalue = 0;

    for(int k = 0; k < Width ; ++k){
        float Mdelement = Md[ty*Width + k];
        float Ndelement = Nd[k*Width + tx];
        Pvalue += (Mdelement*Ndelement);
    }
    Pd[ty*Width + tx] = Pvalue;
}

void MatrixMultiplication(float *M, float *N, float *P, int Width) 
{
    int size = Width*Width*sizeof(float);
    float *Md, *Nd, *Pd;
	int k = 100;
	int l = 100;

	hipEventCreate(&start);
	hipEventCreate(&stop);

    //Transfer M and N to device memory
    hipMalloc((void**)&Md, size);
    hipMemcpy(Md,M,size,hipMemcpyHostToDevice);
    hipMalloc((void**)&Nd, size);
    hipMemcpy(Nd,N,size,hipMemcpyHostToDevice);
    
    //Allocate P on the device
    hipMalloc((void**)&Pd,size);

    //Setup the execution configuration
    dim3 dimBlock((k-1)/Width+1,(l-1)/Width+1);
    dim3 dimGrid(Width,Width);

	hipEventRecord(start, 0);			// use same timing*

    //Launch the device computation threads!
    MatrixMulKernel<<<dimGrid,dimBlock>>>(Md,Nd,Pd,Width);

	//Transfer P from device to host
	hipMemcpy(P,Pd,size,hipMemcpyDeviceToHost);

	hipEventRecord(stop, 0);     		// measure end time
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time_ms, start, stop );

    //Free device matrices
    hipFree(Md);
    hipFree(Nd);
    hipFree(Pd);
}

int main(int argc, char** argv) 
{

    void MatrixMultiplication(float *, float *, float *, int);

	const int Width = atoi(argv[1]);
    float M[Width*Width], N[Width*Width], P[Width*Width];

    for (int i = 0; i < Width; i++) {
        for (int j = 0; j < Width; j++) {
            M[i * Width + j] = i;
            N[i * Width + j] = j;
            P[i * Width + j] = 0;
        }
    }

    MatrixMultiplication(M, N, P, Width);
    for (int i = 0; i < Width; i++) {
        for (int j = 0; j < Width; j++) {
            printf("%f \t", P[i * Width + j]);
        }
        printf("\n")
    }

	printf("Computation time of GPU: %f ms.\n This is a change", elapsed_time_ms);  // exe. time

    return 0;
}
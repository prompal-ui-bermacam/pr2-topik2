/**
*	Author : Rushikesh Gaidhani
*	Topic  : Matrix Multiplication on GPGPU using CUDA
*/

#include<hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
 

hipEvent_t start, stop;  // using cuda events to measure time
float elapsed_time_ms;    // which is applicable for asynchronous code also

//Matrix multiplication kernel - thread specification
__global__ void MatrixMulKernel(float *Md, float *Nd, float *Pd, int Width)
{
    //2D Thread ID
    int tx = threadIdx.x + blockIdx.x*blockDim.x;
    int ty = threadIdx.y + blockIdx.y*blockDim.y;
    
    //Pvalue stores the Pd element that is computed by the thread
    float Pvalue = 0;

    for(int k = 0; k < Width ; ++k){
        float Mdelement = Md[ty*Width + k];
        float Ndelement = Nd[k*Width + tx];
        Pvalue += (Mdelement*Ndelement);
    }
    Pd[ty*Width + tx] = Pvalue;
}

void MatrixMultiplication(float *M, float *N, float *P, int Width) 
{
    int size = Width*Width*sizeof(float);
    float *Md, *Nd, *Pd;
	int k = 100;
	int l = 100;

	hipEventCreate(&start);
	hipEventCreate(&stop);

    //Transfer M and N to device memory
    hipMalloc((void**)&Md, size);
    hipMemcpy(Md,M,size,hipMemcpyHostToDevice);
    hipMalloc((void**)&Nd, size);
    hipMemcpy(Nd,N,size,hipMemcpyHostToDevice);
    
    //Allocate P on the device
    hipMalloc((void**)&Pd,size);

    //Setup the execution configuration
    dim3 dimBlock((k-1)/Width+1,(l-1)/Width+1);
    dim3 dimGrid(Width,Width);

	hipEventRecord(start, 0);			// use same timing*

    //Launch the device computation threads!
    MatrixMulKernel<<<dimGrid,dimBlock>>>(Md,Nd,Pd,Width);

	//Transfer P from device to host
	hipMemcpy(P,Pd,size,hipMemcpyDeviceToHost);

	hipEventRecord(stop, 0);     		// measure end time
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsed_time_ms, start, stop );

    //Free device matrices
    hipFree(Md);
    hipFree(Nd);
    hipFree(Pd);
}

void verify(float *A, float *B, float *C, unsigned int m, unsigned int k, unsigned int n) {
    const float relativeTolerance = 1e-6;

    for(int row = 0; row < m; ++row) {
        for(int col = 0; col < n; ++col) {
            float sum = 0;
            for(unsigned int i = 0; i < k; ++i) {
              sum += A[row*k + i]*B[i*n + col];
            }

            float relativeError = (sum - C[row*n + col])/sum;
            if (relativeError > relativeTolerance
                || relativeError < -relativeTolerance) {
            	printf("(%d, %d) = %f, supposed to be %f\n", row, col, C[row*n + col], sum); 
                printf("TEST FAILED\n\n");
                exit(0);
            }
        }
    }
    printf("TEST PASSED\n\n");
}

int main(int argc, char** argv) 
{

    void MatrixMultiplication(float *, float *, float *, int);

	const int Width = atoi(argv[1]);
    float M[Width*Width], N[Width*Width], P[Width*Width];

    for (int i = 0; i < Width * Width; i++) {
        M[i] = (rand()%100)/100.00;
        N[i] = (rand()%100)/100.00;
        P[i] = 0;
    }

    MatrixMultiplication(M, N, P, Width);
    // for (int i = 0; i < Width; i++) {
    //     for (int j = 0; j < Width; j++) {
    //         printf("%f \t", P[i * Width + j]);
    //     }
    //     printf("\n")
    // }

	printf("Computation time of GPU: %f ms.\n This is a change", elapsed_time_ms);  // exe. time

    verify(M, N, P, Width, Width, Width);

    return 0;
}